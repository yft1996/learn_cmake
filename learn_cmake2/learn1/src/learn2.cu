#include "hip/hip_runtime.h"
#include "learn2.cuh"
#include <cstdio>
__global__ void kernel_hello()
{
    const int x=threadIdx.x+blockIdx.x*blockDim.x;
    const int y=threadIdx.y+blockIdx.y*blockDim.y;
    printf("hello!\tx:%d,y:%d\n",x,y);
}

void hello()
{
    std::cout<<"调用文件："<<__FILE__<<std::endl<<std::endl;
    dim3 hello_block(4,4);
    dim3 hello_grid(2,2);
    kernel_hello<<<hello_grid,hello_block>>>();
    hipDeviceSynchronize();
}
